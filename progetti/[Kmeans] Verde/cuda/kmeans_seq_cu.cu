/*
 * k-Means clustering algorithm
 *
 * CUDA version
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2022/2023
 *
 * Version: 1.0
 *
 * (c) 2022 Diego García-Álvarez, Arturo Gonzalez-Escribano
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define MAXLINE 2000
#define MAXCAD 200

//Macros
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))


double get_walltime() {
    struct timeval time;
    if (gettimeofday(&time, NULL)) {
        // In caso di errore, restituisce 0.
        return 0;
    }
    return (double) time.tv_sec + (double) time.tv_usec * 1e-6;
}

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL( a )	{ \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}
#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}

/*
Function showFileError: It displays the corresponding error during file reading.
*/
void showFileError(int error, char* filename)
{
	printf("Error\n");
	switch (error)
	{
		case -1:
			fprintf(stderr,"\tFile %s has too many columns.\n", filename);
			fprintf(stderr,"\tThe maximum number of columns has been exceeded. MAXLINE: %d.\n", MAXLINE);
			break;
		case -2:
			fprintf(stderr,"Error reading file: %s.\n", filename);
			break;
		case -3:
			fprintf(stderr,"Error writing file: %s.\n", filename);
			break;
	}
	fflush(stderr);
}

void writeTimes(const char* filename, double *returns) {
    FILE *fp = fopen(filename, "a");
    if (fp == NULL) {
        fprintf(stderr, "Error writing to file %s\n", filename);
        return;
    }

    fprintf(fp, "%.6f;%.6f;%.6f;%.0f\n", returns[0], returns[1], returns[2], returns[3]);
    fclose(fp);
}

/*
Function readInput: It reads the file to determine the number of rows and columns.
*/
int readInput(char* filename, int *lines, int *samples)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int contlines, contsamples = 0;

    contlines = 0;

    if ((fp=fopen(filename,"r"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL)
		{
			if (strchr(line, '\n') == NULL)
			{
				return -1;
			}
            contlines++;
            ptr = strtok(line, delim);
            contsamples = 0;
            while(ptr != NULL)
            {
            	contsamples++;
				ptr = strtok(NULL, delim);
	    	}
        }
        fclose(fp);
        *lines = contlines;
        *samples = contsamples;
        return 0;
    }
    else
	{
    	return -2;
	}
}

/*
Function readInput2: It loads data from file.
*/
int readInput2(char* filename, double* data)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int i = 0;

    if ((fp=fopen(filename,"rt"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL)
        {
            ptr = strtok(line, delim);
            while(ptr != NULL)
            {
            	data[i] = atof(ptr);
            	i++;
				ptr = strtok(NULL, delim);
	   		}
	    }
        fclose(fp);
        return 0;
    }
    else
	{
    	return -2; //No file found
	}
}

/*
Function writeResult: It writes in the output file the cluster of each sample (point).
*/
int writeResult(int *classMap, int lines, const char* filename)
{
    FILE *fp;

    if ((fp=fopen(filename,"wt"))!=NULL)
    {
        for(int i=0; i<lines; i++)
        {
        	fprintf(fp,"%d\n",classMap[i]);
        }
        fclose(fp);

        return 0;
    }
    else
	{
    	return -3; //No file found
	}
}

/*

Function initCentroids: This function copies the values of the initial centroids, using their
position in the input data structure as a reference map.
*/
void initCentroids(const double *data, double* centroids, int* centroidPos, int samples, int K)
{
	int i;
	int idx;
	for(i=0; i<K; i++)
	{
		idx = centroidPos[i];
		memcpy(&centroids[i*samples], &data[idx*samples], (samples*sizeof(double)));
	}
}

/*
Function euclideanDistance: Euclidean distance
This function could be modified
*/
double euclideanDistance(double *point, double *center, int samples)
{
	double dist=0.0;
	for(int i=0; i<samples; i++)
	{
		dist+= (point[i]-center[i])*(point[i]-center[i]);
	}
	dist = sqrt(dist);
	return(dist);
}

/*
Function zeroFloatMatriz: Set matrix elements to 0
This function could be modified
*/
void zeroFloatMatriz(double *matrix, int rows, int columns)
{
	int i,j;
	for (i=0; i<rows; i++)
		for (j=0; j<columns; j++)
			matrix[i*columns+j] = 0.0;
}

/*
Function zeroIntArray: Set array elements to 0
This function could be modified
*/
void zeroIntArray(int *array, int size)
{
	int i;
	for (i=0; i<size; i++)
		array[i] = 0;
}



int main(int argc, char* argv[])
{

	//START CLOCK***************************************
	double start, end;
	start = get_walltime();
	//**************************************************
	/*
	* PARAMETERS
	*
	* argv[1]: Input data file
	* argv[2]: Number of clusters
	* argv[3]: Maximum number of iterations of the method. Algorithm termination condition.
	* argv[4]: Minimum percentage of class changes. Algorithm termination condition.
	*          If between one iteration and the next, the percentage of class changes is less than
	*          this percentage, the algorithm stops.
	* argv[5]: Precision in the centroid distance after the update.
	*          It is an algorithm termination condition. If between one iteration of the algorithm
	*          and the next, the maximum distance between centroids is less than this precision, the
	*          algorithm stops.
	* argv[6]: Output file. Class assigned to each point of the input file.
	* */
  if(argc !=  8)
	{
		fprintf(stderr,"EXECUTION ERROR K-MEANS: Parameters are not correct.\n");
		fprintf(stderr,"./KMEANS [Input Filename] [Number of clusters] [Number of iterations] [Number of changes] [Threshold] [Output data file]\n");
		fflush(stderr);
		exit(-1);
	}

	// Reading the input data
	// lines = number of points; samples = number of dimensions per point
	int lines = 0, samples= 0;

	int error = readInput(argv[1], &lines, &samples);
	if(error != 0)
	{
		showFileError(error,argv[1]);
		exit(error);
	}

	double *data = (double*)calloc(lines*samples,sizeof(double));
	if (data == NULL)
	{
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}
	error = readInput2(argv[1], data);
	if(error != 0)
	{
		showFileError(error,argv[1]);
		exit(error);
	}

	// Parameters
	int K=atoi(argv[2]);
	int maxIterations=atoi(argv[3]);
	int minChanges= (int)(lines*atof(argv[4])/100.0);
	double maxThreshold=atof(argv[5]);

	int *centroidPos = (int*)calloc(K,sizeof(int));
	double *centroids = (double*)calloc(K*samples,sizeof(double));
	int *classMap = (int*)calloc(lines,sizeof(int));

  double *returns = (double*)calloc(4,sizeof(double));


    if (centroidPos == NULL || centroids == NULL || classMap == NULL)
	{
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}

	// Initial centrodis
	srand(0);
	int i;
	for(i=0; i<K; i++)
		centroidPos[i]=rand()%lines;

	// Loading the array of initial centroids with the data from the array data
	// The centroids are points stored in the data array.
	initCentroids(data, centroids, centroidPos, samples, K);

/*
	printf("\n\tData file: %s \n\tPoints: %d\n\tDimensions: %d\n", argv[1], lines, samples);
	printf("\tNumber of clusters: %d\n", K);
	printf("\tMaximum number of iterations: %d\n", maxIterations);
	printf("\tMinimum number of changes: %d [%g%% of %d points]\n", minChanges, atof(argv[4]), lines);
	printf("\tMaximum centroid precision: %f\n", maxThreshold);
*/
	//END CLOCK*****************************************
	end = get_walltime();
  returns[0]=end - start;
  // printf("\nMemory allocation: %f seconds\n", end - start);
	fflush(stdout);

	CHECK_CUDA_CALL( hipSetDevice(0) );
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
	//**************************************************
	//START CLOCK***************************************
	start = get_walltime();
	//**************************************************
  //char *outputMsg = (char *)calloc(10000,sizeof(char));
  //char line[100];

	int j;
	int clusterId;
	double dist, minDist;
	int it=0;
	int changes = 0;
	double maxDist;

	//pointPerClass: number of points classified in each class
	//auxCentroids: mean of the points in each class
	int *pointsPerClass = (int *)malloc(K*sizeof(int));
	double *auxCentroids = (double*)malloc(K*samples*sizeof(double));
	double *distCentroids = (double*)malloc(K*sizeof(double));
	if (pointsPerClass == NULL || auxCentroids == NULL || distCentroids == NULL)
	{
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */

	do{
		it++;

		//1. Calculate the distance from each point to the centroid
		//Assign each point to the nearest centroid.
		changes = 0;
		for(i=0; i<lines; i++)
		{
			clusterId=1;
			minDist=FLT_MAX;
			for(j=0; j<K; j++)
			{
				dist=euclideanDistance(&data[i*samples], &centroids[j*samples], samples);

				if(dist < minDist)
				{
					minDist=dist;
					clusterId=j+1;
				}
			}
			if(classMap[i]!=clusterId)
			{
				changes++;
			}
			classMap[i]=clusterId;
		}

		// 2. Recalculates the centroids: calculates the mean within each cluster
		zeroIntArray(pointsPerClass,K);
		zeroFloatMatriz(auxCentroids,K,samples);

		for(i=0; i<lines; i++)
		{
			clusterId=classMap[i];
			pointsPerClass[clusterId-1] = pointsPerClass[clusterId-1] +1;
			for(j=0; j<samples; j++){
				auxCentroids[(clusterId-1)*samples+j] += data[i*samples+j];
			}
		}

		for(i=0; i<K; i++)
		{
			for(j=0; j<samples; j++){
				auxCentroids[i*samples+j] /= pointsPerClass[i];
			}
		}

		maxDist=FLT_MIN;
		for(i=0; i<K; i++){
			distCentroids[i]=euclideanDistance(&centroids[i*samples], &auxCentroids[i*samples], samples);
			if(distCentroids[i]>maxDist) {
				maxDist=distCentroids[i];
			}
		}
		memcpy(centroids, auxCentroids, (K*samples*sizeof(double)));

		//sprintf(line,"\n[%d] Cluster changes: %d\tMax. centroid distance: %f", it, changes, maxDist);
		//outputMsg = strcat(outputMsg,line);

	} while((changes>minChanges) && (it<maxIterations) && (maxDist>maxThreshold));

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */
	// Output and termination conditions
  //printf("%s",outputMsg);

	CHECK_CUDA_CALL( hipDeviceSynchronize() );

	//END CLOCK*****************************************
	end = get_walltime();
	printf("\nComputation: %f seconds", end - start);
  returns[1]=end - start;
  returns[3]= (double)it ;

	fflush(stdout);
	//**************************************************
	//START CLOCK***************************************
	start = get_walltime();
	//**************************************************



	if (changes <= minChanges) {
		printf("\n\nTermination condition:\nMinimum number of changes reached: %d [%d]", changes, minChanges);
	}
	else if (it >= maxIterations) {
		printf("\n\nTermination condition:\nMaximum number of iterations reached: %d [%d]", it, maxIterations);
	}
	else {
		printf("\n\nTermination condition:\nCentroid update precision reached: %g [%g]", maxDist, maxThreshold);
	}

	// Writing the classification of each point to the output file.
	error = writeResult(classMap, lines, argv[6]);
	if(error != 0)
	{
		showFileError(error, argv[6]);
		exit(error);
	}

	//Free memory
	free(data);
	free(classMap);
	free(centroidPos);
	free(centroids);
	free(distCentroids);
	free(pointsPerClass);
	free(auxCentroids);


	//END CLOCK*****************************************
	end = get_walltime();

  returns[2]=end - start;
  // printf("\n\nMemory deallocation: %f seconds\n", end - start);
	fflush(stdout);
  writeTimes(argv[7], returns);
	//***************************************************/
	return 0;
}
